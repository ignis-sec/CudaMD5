
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <stdint.h>

#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/device_functions.h>

#define BLOCKSIZE 512
__device__ void CudaMD5(unsigned char* data, int length, uint32_t* a1, uint32_t* b1, uint32_t* c1, uint32_t* d1);
char* digestMD5(uint32_t hash[4]);
__global__ void getNext(int* iter, uint8_t* result, uint32_t* hash, uint8_t *solbuf, uint32_t* solhash);
__device__ static const char allowed_characters[] = "0123456789abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ";
__device__ static const int alphabet_length = 62;
__device__ static const char salt[] = "ignisET1Y";
__device__ static const int saltlen = 9;
__device__ static const int MAX_UNHASHED_LEN = 32;

int main(void) {
	char* msg;
	int cudaStatus;
	uint8_t *plain;
	uint8_t *d_plain;
	uint32_t *hash;
	uint32_t *d_hash;
	uint8_t* solbuf;
	uint8_t* d_solbuf;
	uint32_t* solhash;
	uint32_t* d_solhash;
	int* iter;
	plain = (uint8_t*)malloc(BLOCKSIZE * 32);
	hash = (uint32_t*)malloc(BLOCKSIZE * 4 * sizeof(uint32_t));
	solbuf = (uint8_t*)malloc(32);
	solhash = (uint32_t*)malloc(4 * sizeof(uint32_t));
	hipMalloc((void**)&d_plain, 32 * BLOCKSIZE);
	hipMalloc((void**)&d_hash, 4 * BLOCKSIZE * sizeof(uint32_t));
	hipMalloc((void**)&d_solbuf, 32);
	hipMalloc((void**)&d_solhash, 4 * sizeof(uint32_t));
	for (int i = 0; i < 78125; i++) {//78125
		 hipMalloc((void**)&iter, sizeof(uint32_t));
		 hipMemcpy(iter,&i, sizeof(uint32_t),hipMemcpyHostToDevice);
		 getNext<<<512,1>>>(iter,d_plain, d_hash, d_solbuf, d_solhash);
		 hipMemcpy(plain, d_plain, 32 * BLOCKSIZE, hipMemcpyDeviceToHost);
		 hipMemcpy(hash, d_hash, 4 * sizeof(uint32_t) * BLOCKSIZE, hipMemcpyDeviceToHost);
		 hipMemcpy(solbuf, d_solbuf, 32, hipMemcpyDeviceToHost);
		 hipMemcpy(solhash, d_solhash, 4 * sizeof(uint32_t), hipMemcpyDeviceToHost);
		 for (int j = 0; j < BLOCKSIZE; j++) {
			 char* digest = digestMD5(&hash[4*j]);
			 char* digest2 = digestMD5(solhash);
			 printf("%5d %16s: %32s; solbuf:%s, solhash:%s\n",i*BLOCKSIZE, &plain[32*j], digest, solbuf, digest2);

		 }
		 hipFree(iter);

	}

	return 0;
}

char* digestMD5(uint32_t hash[4]) {
	char* digest;
	digest = (char*)malloc(33);
	for (int j = 0; j < 4; j++) {
		uint8_t bytes[4];
		bytes[0] = (uint8_t)hash[j];
		bytes[1] = (uint8_t)(hash[j] >> 8);
		bytes[2] = (uint8_t)(hash[j] >> 16);
		bytes[3] = (uint8_t)(hash[j] >> 24);
		for(int i=0;i<4;i++)
			sprintf(&digest[2 * (4*j+i)], "%02X", bytes[i]);
	}	
	digest[32] = '\0';
	return digest;
}


__global__ void getNext(int* iter, uint8_t* result, uint32_t* hash, uint8_t* solbuf, uint32_t* solhash) {
	int _offset;
	_offset = *iter*BLOCKSIZE + blockIdx.x;
	char* extension;
	extension = (char*)malloc(MAX_UNHASHED_LEN - saltlen);
	memcpy(extension, "\0", MAX_UNHASHED_LEN - saltlen);
	int maxi = 0;
	
	
	for (int i = 0; i < 32; i++) {
		int rem = _offset % alphabet_length;
		int div = _offset / alphabet_length;
		_offset = div;
		//printf("rem: %d div:%d\n", rem, div);
		extension[i] = allowed_characters[rem];
		maxi = i;
		if (div == 0) {
			break;
		}
	}

	unsigned char* hashin;
	hashin = (unsigned char* )malloc(saltlen + maxi);

	memcpy(&result[blockIdx.x*32], salt, saltlen);
	memcpy(&result[blockIdx.x*32] + saltlen, extension, MAX_UNHASHED_LEN - saltlen);
	memcpy(hashin, &result[blockIdx.x*32], 32);
	CudaMD5(hashin,maxi+saltlen+1, &hash[blockIdx.x*4], &hash[blockIdx.x*4+1], &hash[blockIdx.x*4+2], &hash[blockIdx.x*4+3]);
	uint8_t tb;
	uint32_t tw;
	int digestedCounter = 0;
	int bMatchFlag = 1;
	for (int i = 0; i < 4; i++) {
		if (!bMatchFlag)break;
		tw = hash[blockIdx.x * 4 + i];
		for (int j = 0; j < 4; j++) {
			if (!bMatchFlag)break;
			for (int k = 0; k < 2; k++) {
				tb = tw << 8 * j;
				if (k == 0) {
					tb = tb & 0xf0;
				}
				else {
					tb = (tb & 0x0f)/8;
				}
				if (digestedCounter == 0 && tb!= 0) {
					bMatchFlag = false;
				}else if (digestedCounter == 1 && tb != 14) { //e
					bMatchFlag = false;
				}
				else if (tb > 9) {
					bMatchFlag = false;
				}
				digestedCounter++;
			}
		}
	}
	if (bMatchFlag) {
		memcpy(solbuf, hashin, 32);
		memcpy(solhash, &hash[blockIdx.x * 4], 4* sizeof(uint32_t));
	}
	free(extension);
	free(hashin);
}


















/* F, G and H are basic MD5 functions: selection, majority, parity */
#define F(x, y, z) (((x) & (y)) | ((~x) & (z)))
#define G(x, y, z) (((x) & (z)) | ((y) & (~z)))
#define H(x, y, z) ((x) ^ (y) ^ (z))
#define I(x, y, z) ((y) ^ ((x) | (~z)))

/* ROTATE_LEFT rotates x left n bits */
#define ROTATE_LEFT(x, n) (((x) << (n)) | ((x) >> (32-(n))))

/* FF, GG, HH, and II transformations for rounds 1, 2, 3, and 4 */
/* Rotation is separate from addition to prevent recomputation */
#define FF(a, b, c, d, x, s, ac) \
  {(a) += F ((b), (c), (d)) + (x) + (uint32_t)(ac); \
    (a) = ROTATE_LEFT ((a), (s)); \
    (a) += (b); \
  }
#define GG(a, b, c, d, x, s, ac) \
  {(a) += G ((b), (c), (d)) + (x) + (uint32_t)(ac); \
    (a) = ROTATE_LEFT ((a), (s)); \
    (a) += (b); \
  }
#define HH(a, b, c, d, x, s, ac) \
  {(a) += H ((b), (c), (d)) + (x) + (uint32_t)(ac); \
    (a) = ROTATE_LEFT ((a), (s)); \
    (a) += (b); \
  }
#define II(a, b, c, d, x, s, ac) \
  {(a) += I ((b), (c), (d)) + (x) + (uint32_t)(ac); \
    (a) = ROTATE_LEFT ((a), (s)); \
    (a) += (b); \
  }

__device__ void CudaMD5(unsigned char* data, int length, uint32_t* a1, uint32_t* b1, uint32_t* c1, uint32_t* d1) {
	const uint32_t a0 = 0x67452301;
	const uint32_t b0 = 0xEFCDAB89;
	const uint32_t c0 = 0x98BADCFE;
	const uint32_t d0 = 0x10325476;

	uint32_t a = 0;
	uint32_t b = 0;
	uint32_t c = 0;
	uint32_t d = 0;

	uint32_t vals[14] = { 0,0,0,0,0,0,0,0,0,0,0,0,0,0 };

	int i = 0;
	for (i = 0; i < length; i++) {
		vals[i / 4] |= data[i] << ((i % 4) * 8);
	}

	vals[i / 4] |= 0x80 << ((i % 4) * 8);

	uint32_t bitlen = length * 8;

#define in0  (vals[0])//x
#define in1  (vals[1])//y
#define in2  (vals[2])//z
#define in3  (vals[3])
#define in4  (vals[4])
#define in5  (vals[5])
#define in6  (vals[6])
#define in7  (vals[7])
#define in8  (vals[8])
#define in9  (vals[9])
#define in10 (vals[10])
#define in11 (vals[11])
#define in12 (vals[12])
#define in13 (vals[13])
#define in14 (bitlen) //w = bit length
#define in15 (0)

	//Initialize hash value for this chunk:
	a = a0;
	b = b0;
	c = c0;
	d = d0;

	/* Round 1 */
#define S11 7
#define S12 12
#define S13 17
#define S14 22
	FF(a, b, c, d, in0, S11, 3614090360); /* 1 */
	FF(d, a, b, c, in1, S12, 3905402710); /* 2 */
	FF(c, d, a, b, in2, S13, 606105819); /* 3 */
	FF(b, c, d, a, in3, S14, 3250441966); /* 4 */
	FF(a, b, c, d, in4, S11, 4118548399); /* 5 */
	FF(d, a, b, c, in5, S12, 1200080426); /* 6 */
	FF(c, d, a, b, in6, S13, 2821735955); /* 7 */
	FF(b, c, d, a, in7, S14, 4249261313); /* 8 */
	FF(a, b, c, d, in8, S11, 1770035416); /* 9 */
	FF(d, a, b, c, in9, S12, 2336552879); /* 10 */
	FF(c, d, a, b, in10, S13, 4294925233); /* 11 */
	FF(b, c, d, a, in11, S14, 2304563134); /* 12 */
	FF(a, b, c, d, in12, S11, 1804603682); /* 13 */
	FF(d, a, b, c, in13, S12, 4254626195); /* 14 */
	FF(c, d, a, b, in14, S13, 2792965006); /* 15 */
	FF(b, c, d, a, in15, S14, 1236535329); /* 16 */

	/* Round 2 */
#define S21 5
#define S22 9
#define S23 14
#define S24 20
	GG(a, b, c, d, in1, S21, 4129170786); /* 17 */
	GG(d, a, b, c, in6, S22, 3225465664); /* 18 */
	GG(c, d, a, b, in11, S23, 643717713); /* 19 */
	GG(b, c, d, a, in0, S24, 3921069994); /* 20 */
	GG(a, b, c, d, in5, S21, 3593408605); /* 21 */
	GG(d, a, b, c, in10, S22, 38016083); /* 22 */
	GG(c, d, a, b, in15, S23, 3634488961); /* 23 */
	GG(b, c, d, a, in4, S24, 3889429448); /* 24 */
	GG(a, b, c, d, in9, S21, 568446438); /* 25 */
	GG(d, a, b, c, in14, S22, 3275163606); /* 26 */
	GG(c, d, a, b, in3, S23, 4107603335); /* 27 */
	GG(b, c, d, a, in8, S24, 1163531501); /* 28 */
	GG(a, b, c, d, in13, S21, 2850285829); /* 29 */
	GG(d, a, b, c, in2, S22, 4243563512); /* 30 */
	GG(c, d, a, b, in7, S23, 1735328473); /* 31 */
	GG(b, c, d, a, in12, S24, 2368359562); /* 32 */

	/* Round 3 */
#define S31 4
#define S32 11
#define S33 16
#define S34 23
	HH(a, b, c, d, in5, S31, 4294588738); /* 33 */
	HH(d, a, b, c, in8, S32, 2272392833); /* 34 */
	HH(c, d, a, b, in11, S33, 1839030562); /* 35 */
	HH(b, c, d, a, in14, S34, 4259657740); /* 36 */
	HH(a, b, c, d, in1, S31, 2763975236); /* 37 */
	HH(d, a, b, c, in4, S32, 1272893353); /* 38 */
	HH(c, d, a, b, in7, S33, 4139469664); /* 39 */
	HH(b, c, d, a, in10, S34, 3200236656); /* 40 */
	HH(a, b, c, d, in13, S31, 681279174); /* 41 */
	HH(d, a, b, c, in0, S32, 3936430074); /* 42 */
	HH(c, d, a, b, in3, S33, 3572445317); /* 43 */
	HH(b, c, d, a, in6, S34, 76029189); /* 44 */
	HH(a, b, c, d, in9, S31, 3654602809); /* 45 */
	HH(d, a, b, c, in12, S32, 3873151461); /* 46 */
	HH(c, d, a, b, in15, S33, 530742520); /* 47 */
	HH(b, c, d, a, in2, S34, 3299628645); /* 48 */

	/* Round 4 */
#define S41 6
#define S42 10
#define S43 15
#define S44 21
	II(a, b, c, d, in0, S41, 4096336452); /* 49 */
	II(d, a, b, c, in7, S42, 1126891415); /* 50 */
	II(c, d, a, b, in14, S43, 2878612391); /* 51 */
	II(b, c, d, a, in5, S44, 4237533241); /* 52 */
	II(a, b, c, d, in12, S41, 1700485571); /* 53 */
	II(d, a, b, c, in3, S42, 2399980690); /* 54 */
	II(c, d, a, b, in10, S43, 4293915773); /* 55 */
	II(b, c, d, a, in1, S44, 2240044497); /* 56 */
	II(a, b, c, d, in8, S41, 1873313359); /* 57 */
	II(d, a, b, c, in15, S42, 4264355552); /* 58 */
	II(c, d, a, b, in6, S43, 2734768916); /* 59 */
	II(b, c, d, a, in13, S44, 1309151649); /* 60 */
	II(a, b, c, d, in4, S41, 4149444226); /* 61 */
	II(d, a, b, c, in11, S42, 3174756917); /* 62 */
	II(c, d, a, b, in2, S43, 718787259); /* 63 */
	II(b, c, d, a, in9, S44, 3951481745); /* 64 */

	a += a0;
	b += b0;
	c += c0;
	d += d0;

	*a1 = a;
	*b1 = b;
	*c1 = c;
	*d1 = d;
}



